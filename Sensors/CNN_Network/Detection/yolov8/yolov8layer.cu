#include "hip/hip_runtime.h"
//
// From https://github.com/wang-xinyu/tensorrtx
//
#include "yolov8layer.h"
#include <assert.h>
#include <math.h>
#include <vector>
#include <iostream>
#include "../../utils/cuda_utils.h"

namespace Tn {
    template<typename T>
    void write(char*& buffer, const T& val) {
        *reinterpret_cast<T*>(buffer) = val;
        buffer += sizeof(T);
    }

    template<typename T>
    void read(const char*& buffer, T& val) {
        val = *reinterpret_cast<const T*>(buffer);
        buffer += sizeof(T);
    }
}  // namespace Tn


namespace nvinfer1 {
    Yolov8LayerPlugin::Yolov8LayerPlugin(int classCount, int netWidth, int netHeight, int maxOut) {
    mClassCount = classCount;
    mYoloV8NetWidth = netWidth;
    mYoloV8netHeight = netHeight;
    mMaxOutObject = maxOut;
}

Yolov8LayerPlugin::~Yolov8LayerPlugin() {}

Yolov8LayerPlugin::Yolov8LayerPlugin(const void* data, size_t length) {
    using namespace Tn;
    const char* d = reinterpret_cast<const char*>(data), * a = d;
    read(d, mClassCount);
    read(d, mThreadCount);
    read(d, mYoloV8NetWidth);
    read(d, mYoloV8netHeight);
    read(d, mMaxOutObject);

    assert(d == a + length);
}

void Yolov8LayerPlugin::serialize(void* buffer) const TRT_NOEXCEPT {

    using namespace Tn;
    char* d = static_cast<char*>(buffer), * a = d;
    write(d, mClassCount);
    write(d, mThreadCount);
    write(d, mYoloV8NetWidth);
    write(d, mYoloV8netHeight);
    write(d, mMaxOutObject);

    assert(d == a + getSerializationSize());
}

size_t Yolov8LayerPlugin::getSerializationSize() const TRT_NOEXCEPT {
    return sizeof(mClassCount) + sizeof(mThreadCount) + sizeof(mYoloV8netHeight) + sizeof(mYoloV8NetWidth) + sizeof(mMaxOutObject);
}

int Yolov8LayerPlugin::initialize() TRT_NOEXCEPT {
    return 0;
}

nvinfer1::Dims Yolov8LayerPlugin::getOutputDimensions(int index, const nvinfer1::Dims* inputs, int nbInputDims) TRT_NOEXCEPT {
    int total_size = mMaxOutObject * sizeof(Yolo::Detection) / sizeof(float);
    return nvinfer1::Dims3(total_size + 1, 1, 1);
}

void Yolov8LayerPlugin::setPluginNamespace(const char* pluginNamespace) TRT_NOEXCEPT {
    mPluginNamespace = pluginNamespace;
}

const char* Yolov8LayerPlugin::getPluginNamespace() const TRT_NOEXCEPT {
    return mPluginNamespace;
}

nvinfer1::DataType Yolov8LayerPlugin::getOutputDataType(int index, const nvinfer1::DataType* inputTypes, int nbInputs) const TRT_NOEXCEPT {
    return nvinfer1::DataType::kFLOAT;
}

bool Yolov8LayerPlugin::isOutputBroadcastAcrossBatch(int outputIndex, const bool* inputIsBroadcasted, int nbInputs) const TRT_NOEXCEPT {

    return false;
}

bool Yolov8LayerPlugin::canBroadcastInputAcrossBatch(int inputIndex) const TRT_NOEXCEPT {

    return false;
}

void Yolov8LayerPlugin::configurePlugin(nvinfer1::PluginTensorDesc const* in, int nbInput, nvinfer1::PluginTensorDesc const* out, int nbOutput) TRT_NOEXCEPT {};

void Yolov8LayerPlugin::attachToContext(cudnnContext* cudnnContext, cublasContext* cublasContext, IGpuAllocator* gpuAllocator) TRT_NOEXCEPT {};

void Yolov8LayerPlugin::detachFromContext() TRT_NOEXCEPT {}

const char* Yolov8LayerPlugin::getPluginType() const TRT_NOEXCEPT {

    return "Yolov8Layer_TRT";
}

const char* Yolov8LayerPlugin::getPluginVersion() const TRT_NOEXCEPT {
    return "1";
}

void Yolov8LayerPlugin::destroy() TRT_NOEXCEPT {

    delete this;
}

nvinfer1::IPluginV2IOExt* Yolov8LayerPlugin::clone() const TRT_NOEXCEPT {

    Yolov8LayerPlugin* p = new Yolov8LayerPlugin(mClassCount, mYoloV8NetWidth, mYoloV8netHeight, mMaxOutObject);
    p->setPluginNamespace(mPluginNamespace);
    return p;
}

int Yolov8LayerPlugin::enqueue(int batchSize, const void* TRT_CONST_ENQUEUE* inputs, void* const* outputs, void* workspace, hipStream_t stream) TRT_NOEXCEPT {

    forwardGpu((const float* const*)inputs, (float*)outputs[0], stream, mYoloV8netHeight, mYoloV8NetWidth, batchSize);
    return 0;
}


__device__ float LogistV8(float data) { return 1.0f / (1.0f + expf(-data)); };

__global__ void CalDetectionV8(const float* input, float* output, int numElements, int maxoutobject,
                             const int grid_h, int grid_w, const int stride, int classes, int outputElem) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= numElements) return;

    int total_grid = grid_h * grid_w;
    int info_len = 4 + classes;
    int batchIdx = idx / total_grid;
    int elemIdx = idx % total_grid;
    const float* curInput = input + batchIdx * total_grid * info_len;
    int outputIdx = batchIdx * outputElem;

    int class_id = 0;
    float max_cls_prob = 0.0;
    for (int i = 4; i < info_len; i++) {
        float p = LogistV8(curInput[elemIdx + i * total_grid]);
        if (p > max_cls_prob) {
            max_cls_prob = p;
            class_id = i - 4;
        }
    }

    if (max_cls_prob < 0.1) return;

    int count = (int)atomicAdd(output + outputIdx, 1);
    if (count >= maxoutobject) return;
    char* data = (char*)(output + outputIdx) + sizeof(float) + count * sizeof(Yolo::Detection);
    Yolo::Detection* det = (Yolo::Detection*)(data);

    int row = elemIdx / grid_w;
    int col = elemIdx % grid_w;

    det->conf = max_cls_prob;
    det->class_id = class_id;
    det->bbox[0] = (col + 0.5f - curInput[elemIdx + 0 * total_grid]) * stride;
    det->bbox[1] = (row + 0.5f - curInput[elemIdx + 1 * total_grid]) * stride;
    det->bbox[2] = (col + 0.5f + curInput[elemIdx + 2 * total_grid]) * stride;
    det->bbox[3] = (row + 0.5f + curInput[elemIdx + 3 * total_grid]) * stride;
}

void Yolov8LayerPlugin::forwardGpu(const float* const* inputs, float* output, hipStream_t stream, int mYoloV8netHeight,int mYoloV8NetWidth, int batchSize) {
    int outputElem = 1 + mMaxOutObject * sizeof(Yolo::Detection) / sizeof(float);
    hipMemsetAsync(output, 0, sizeof(float), stream);
    for (int idx = 0; idx < batchSize; ++idx) {
        CUDA_CHECK(hipMemsetAsync(output + idx * outputElem, 0, sizeof(float), stream));
    }
    int numElem = 0;
    int grids[3][2] = { {mYoloV8netHeight / 8, mYoloV8NetWidth / 8}, {mYoloV8netHeight / 16, mYoloV8NetWidth / 16}, {mYoloV8netHeight / 32, mYoloV8NetWidth / 32} };
    int strides[] = { 8, 16, 32 };
    for (unsigned int i = 0; i < 3; i++) {
        int grid_h = grids[i][0];
        int grid_w = grids[i][1];
        int stride = strides[i];
        numElem = grid_h * grid_w * batchSize;
        if (numElem < mThreadCount) mThreadCount = numElem;

        CalDetectionV8 <<<(numElem + mThreadCount - 1) / mThreadCount, mThreadCount, 0, stream >>>
            (inputs[i], output, numElem, mMaxOutObject, grid_h, grid_w, stride, mClassCount, outputElem);
    }
}

PluginFieldCollection YoloV8PluginCreator::mFC{};
std::vector<PluginField> YoloV8PluginCreator::mPluginAttributes;

YoloV8PluginCreator::YoloV8PluginCreator() {
    mPluginAttributes.clear();
    mFC.nbFields = mPluginAttributes.size();
    mFC.fields = mPluginAttributes.data();
}

const char* YoloV8PluginCreator::getPluginName() const TRT_NOEXCEPT {
    return "Yolov8Layer_TRT";
}

const char* YoloV8PluginCreator::getPluginVersion() const TRT_NOEXCEPT {
    return "1";
}

const PluginFieldCollection* YoloV8PluginCreator::getFieldNames() TRT_NOEXCEPT {
    return &mFC;
}

IPluginV2IOExt* YoloV8PluginCreator::createPlugin(const char* name, const PluginFieldCollection* fc) TRT_NOEXCEPT {
    assert(fc->nbFields == 1);
    assert(strcmp(fc->fields[0].name, "netinfo") == 0);
    int* p_netinfo = (int*)(fc->fields[0].data);
    int class_count = p_netinfo[0];
    int input_w = p_netinfo[1];
    int input_h = p_netinfo[2];
    int max_output_object_count = p_netinfo[3];
    Yolov8LayerPlugin* obj = new Yolov8LayerPlugin(class_count, input_w, input_h, max_output_object_count);
    obj->setPluginNamespace(mNamespace.c_str());
    return obj;
}

IPluginV2IOExt* YoloV8PluginCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength) TRT_NOEXCEPT {
    // This object will be deleted when the network is destroyed, which will
    // call YoloV8PluginCreator::destroy()
    Yolov8LayerPlugin* obj = new Yolov8LayerPlugin(serialData, serialLength);
    obj->setPluginNamespace(mNamespace.c_str());
    return obj;
}

} // namespace nvinfer1
