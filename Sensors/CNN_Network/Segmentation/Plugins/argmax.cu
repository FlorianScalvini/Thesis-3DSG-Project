#include "hip/hip_runtime.h"
#include <assert.h>
#include <vector>
#include <iostream>
#include "argmax.h"
#include "../../utils/cuda_utils.h"

namespace Tn
{
    template<typename T>
    void write(char*& buffer, const T& val)
    {
        *reinterpret_cast<T*>(buffer) = val;
        buffer += sizeof(T);
    }

    template<typename T>
    void read(const char*& buffer, T& val)
    {
        val = *reinterpret_cast<const T*>(buffer);
        buffer += sizeof(T);
    }
}

namespace nvinfer1
{
    ArgMaxPlugin::ArgMaxPlugin(int netChannels, int netHeight, int netWidth, int axis)
    {
        mChannels = netChannels;
        mWidth = netWidth;
        mHeight = netHeight;
        mAxis = axis;
    }

    ArgMaxPlugin::~ArgMaxPlugin() {}

    // create the plugin at runtime from a byte stream
    ArgMaxPlugin::ArgMaxPlugin(const void* data, size_t length)
    {
        using namespace Tn;
        const char *d = reinterpret_cast<const char *>(data), *a = d;
        read(d, mChannels);
        read(d, mWidth);
        read(d, mHeight);
        read(d, mAxis);
        read(d, mThreadCount);
        assert(d == a + length);
    }

    void ArgMaxPlugin::serialize(void* buffer) const TRT_NOEXCEPT
    {
        using namespace Tn;
        char* d = static_cast<char*>(buffer), *a = d;
        write(d, mChannels);
        write(d, mWidth);
        write(d, mHeight);
        write(d, mAxis);
        write(d, mThreadCount);
        assert(d == a + getSerializationSize());
    }

    size_t ArgMaxPlugin::getSerializationSize() const TRT_NOEXCEPT
    {
        return sizeof(mChannels) + sizeof(mWidth)  + sizeof(mHeight)  + sizeof(mAxis)  + sizeof(mThreadCount);
    }

    int ArgMaxPlugin::initialize() TRT_NOEXCEPT
    {
        return 0;
    }

    Dims ArgMaxPlugin::getOutputDimensions(int index, const Dims* inputs, int nbInputDims) TRT_NOEXCEPT
    {
        Dims3 outputDims = {mChannels, mHeight, mWidth};
        outputDims.d[mAxis] = 1;
        return outputDims;
    }

    // Set plugin namespace
    void ArgMaxPlugin::setPluginNamespace(const char* pluginNamespace) TRT_NOEXCEPT
    {
        mPluginNamespace = pluginNamespace;
    }

    const char* ArgMaxPlugin::getPluginNamespace() const TRT_NOEXCEPT
    {
        return mPluginNamespace;
    }

    // Return the DataType of the plugin output at the requested index
    DataType ArgMaxPlugin::getOutputDataType(int index, const nvinfer1::DataType* inputTypes, int nbInputs) const TRT_NOEXCEPT
    {
        return DataType::kFLOAT;
    }

    // Return true if output tensor is broadcast across a batch.
    bool ArgMaxPlugin::isOutputBroadcastAcrossBatch(int outputIndex, const bool* inputIsBroadcasted, int nbInputs) const TRT_NOEXCEPT
    {
        return false;
    }

    // Return true if plugin can use input that is broadcast across batch without replication.
    bool ArgMaxPlugin::canBroadcastInputAcrossBatch(int inputIndex) const TRT_NOEXCEPT
    {
        return false;
    }

    // Attach the plugin object to an execution context and grant the plugin the access to some context resource.
    void ArgMaxPlugin::attachToContext(cudnnContext* cudnnContext, cublasContext* cublasContext, IGpuAllocator* gpuAllocator) TRT_NOEXCEPT
    {
    }

    // Detach the plugin object from its execution context.
    void ArgMaxPlugin::detachFromContext() TRT_NOEXCEPT {}

    const char* ArgMaxPlugin::getPluginType() const TRT_NOEXCEPT
    {
        return "ArgMax_TRT";
    }

    const char* ArgMaxPlugin::getPluginVersion() const TRT_NOEXCEPT
    {
        return "1";
    }

    void ArgMaxPlugin::destroy() TRT_NOEXCEPT
    {
        delete this;
    }

    // Clone the plugin
    IPluginV2IOExt* ArgMaxPlugin::clone() const TRT_NOEXCEPT
    {
        ArgMaxPlugin* p = new ArgMaxPlugin(mChannels, mHeight, mWidth, mAxis);
        p->setPluginNamespace(mPluginNamespace);
        return p;
    }

    __global__ void argMaxCuda(const float *input, float* output, int axis, int C, int W, int H)
    {
        int maxIndex = -1;
        float maxValue = -INFINITY;
        unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
        unsigned int b, c, h, w, remaining, outputIndex, index;
        if(axis == 1)
        {
            b = tid / (C * W);
            remaining = tid % (C * W);
            c = remaining / W;
            w = remaining % W;
            outputIndex = b * C * W + c * W + w;
            for (int w = 0; w < W; c++)
            {

                index = b * C * W * H + c * H * W + h * W + w;
                float value = input[index];

                if (value > maxValue)
                {
                    maxValue = value;
                    maxIndex = w;
                }
            }
        }
        else if(axis == 2)
        {
            b = tid / (C * H);
            remaining = tid % (C * H);
            c = remaining / H;
            h = remaining % H;
            outputIndex = b * C * H + c * H + h;
            for (int w = 0; w < W; c++)
            {

                index = b * C * W * H + c * H * W + h * W + w;
                float value = input[index];

                if (value > maxValue)
                {
                    maxValue = value;
                    maxIndex = w;
                }
            }
        }
        else
        {
            b = tid / (W * H);
            remaining = tid % (W * H);
            h = remaining / W;
            w = remaining % W;
            outputIndex = b * H * W + h * W + w;
            for (int c = 0; c < C; c++)
            {

                index = b * C * W * H + c * H * W + h * W + w;
                float value = input[index];

                if (value > maxValue)
                {
                    maxValue = value;
                    maxIndex = c;
                }
            }
        }
        output[outputIndex] = maxIndex;

    }

    void ArgMaxPlugin::forwardGpu(const float* inputs, float *output, hipStream_t stream, int batchSize) {
        int outputElem;
        int block_size;
        if(mAxis == 0)
        {
            outputElem = mWidth * mHeight;
            block_size = (outputElem * batchSize + mThreadCount - 1) / mThreadCount;
        }
        else if(mAxis == 1)
        {
            outputElem = mWidth * mChannels;
            block_size = (outputElem * batchSize + mThreadCount - 1) / mThreadCount;
        }
        else
        {
            outputElem = mChannels * mHeight;
            block_size = (outputElem * batchSize + mThreadCount - 1) / mThreadCount;
        }
        for (int idx = 0; idx < batchSize; ++idx) {
            CUDA_CHECK(hipMemsetAsync(output + idx * outputElem, 0, sizeof(float), stream));
        }
        argMaxCuda<<<block_size, mThreadCount, 0, stream>>>(inputs, output, mAxis, mChannels, mWidth, mHeight);


    }

    int ArgMaxPlugin::enqueue(int batchSize, const void* const* inputs, void* TRT_CONST_ENQUEUE* outputs, void* workspace, hipStream_t stream) TRT_NOEXCEPT
    {
        const float* inputData = static_cast<const float*>(inputs[0]);
        float* outputData = static_cast<float*>(outputs[0]);
        int outputElem;
        int block_size;
        if(mAxis == 0)
        {
            outputElem = mWidth * mHeight;
            block_size = (outputElem * batchSize + mThreadCount - 1) / mThreadCount;
        }
        else if(mAxis == 1)
        {
            outputElem = mWidth * mChannels;
            block_size = (outputElem * batchSize + mThreadCount - 1) / mThreadCount;
        }
        else
        {
            outputElem = mChannels * mHeight;
            block_size = (outputElem * batchSize + mThreadCount - 1) / mThreadCount;
        }
        for (int idx = 0; idx < batchSize; ++idx) {
            CUDA_CHECK(hipMemsetAsync(outputData + idx * outputElem, 0, sizeof(float), stream));
        }
        argMaxCuda<<<block_size, mThreadCount, 0, stream>>>(inputData, outputData, mAxis, mChannels, mWidth, mHeight);
        hipDeviceSynchronize();
        return 0;
    }

    PluginFieldCollection ArgMaxPluginCreator::mFC{};
    std::vector<PluginField> ArgMaxPluginCreator::mPluginAttributes;

    ArgMaxPluginCreator::ArgMaxPluginCreator()
    {
        mPluginAttributes.clear();
        mFC.nbFields = mPluginAttributes.size();
        mFC.fields = mPluginAttributes.data();
    }

    const char* ArgMaxPluginCreator::getPluginName() const TRT_NOEXCEPT
    {
        return "ArgMax_TRT";
    }

    const char* ArgMaxPluginCreator::getPluginVersion() const TRT_NOEXCEPT
    {
        return "1";
    }

    const PluginFieldCollection* ArgMaxPluginCreator::getFieldNames() TRT_NOEXCEPT
    {
        return &mFC;
    }

    IPluginV2IOExt* ArgMaxPluginCreator::createPlugin(const char* name, const PluginFieldCollection* fc) TRT_NOEXCEPT
    {
        assert(fc->nbFields == 1);
        assert(strcmp(fc->fields[0].name, "netinfo") == 0);
        int *p_netinfo = (int*)(fc->fields[0].data);

        ArgMaxPlugin* obj = new ArgMaxPlugin(p_netinfo[0], p_netinfo[1], p_netinfo[2], p_netinfo[3]);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

    IPluginV2IOExt* ArgMaxPluginCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength) TRT_NOEXCEPT
    {
        // This object will be deleted when the network is destroyed, which will
        // call YoloLayerPlugin::destroy()
        ArgMaxPlugin* obj = new ArgMaxPlugin(serialData, serialLength);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }
}

